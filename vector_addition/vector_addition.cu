#include "hip/hip_runtime.h"
// MP 1
#include <wb.h>

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  //ensure that ceil will not raise wrong mem. accesss (boundary test)
  if(i<len) out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here

  size_t sizeVars = inputLength * sizeof(float);

  wbCheck(hipMalloc((void**) &deviceInput1, sizeVars));
  wbCheck(hipMalloc((void**) &deviceInput2, sizeVars));
  wbCheck(hipMalloc((void**) &deviceOutput, sizeVars));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here

  hipMemcpy(deviceInput1, hostInput1 , sizeVars, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2 , sizeVars, hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid((inputLength-1)/256 + 1, 1, 1); //nro of blocks in grid (at least 1 blocks)
  dim3 DimBlock(256, 1, 1);          //nro of threads in blocks

  //What would be the best grid and block sizes to start a kernel call?

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<DimGrid, DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, sizeVars);  

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here

  hipMemcpy(hostOutput, deviceOutput, sizeVars, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here

  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}